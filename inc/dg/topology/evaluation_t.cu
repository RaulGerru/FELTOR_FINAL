#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "dg/blas.h"
#include "dg/functors.h"

#include "evaluation.h"
#include "weights.h"

struct exp_function{
DG_DEVICE
double operator()( double x)
{
    return exp(x);
}
};
struct sin_function{
DG_DEVICE
double operator()( double x)
{
    return sin(x);
}
};

template<class T>
T function( T x, T y)
{
        return exp(x)*exp(y);
}
double function3d( double x, double y, double z)
{
        return exp(x)*exp(y)*exp(z);
}

int main()
{
    std::cout << "This program tests the exblas::dot function. The tests succeed only if the evaluation and grid functions but also the weights and especially the exblas::dot function are correctly implemented and compiled. Furthermore, the compiler implementation of the exp function in the math library must be consistent across platforms to get reproducible results\n";
    std::cout << "A TEST is PASSED if the number in the second column shows EXACTLY 0!\n";
    unsigned n = 3, Nx = 12, Ny = 28, Nz = 100;
    std::cout << "On Grid "<<n<<" x "<<Nx<<" x "<<Ny<<" x "<<Nz<<"\n";

    dg::Grid1d g1d( 1, 2, n, Nx);
    dg::Grid2d g2d( 1, 2, 3, 4, n, Nx, Ny);
    dg::RealGrid2d<float> gf2d( 1, 2, 3, 4, n, Nx, Ny);
    dg::Grid3d g3d( 1, 2, 3, 4, 5, 6, n, Nx, Ny, Nz,dg::PER,dg::PER,dg::PER);

    //test evaluation functions
    const dg::HVec func1d = dg::construct<dg::HVec>( dg::evaluate( exp, g1d));
    const dg::DVec func2d = dg::construct<dg::DVec>( dg::evaluate( function<double>, g2d));
    const dg::fDVec funcf2d = dg::construct<dg::fDVec>( dg::evaluate( function<float>, gf2d));
    const dg::DVec func3d = dg::construct<dg::DVec>( dg::evaluate( function3d, g3d));
    const dg::HVec w1d = dg::construct<dg::HVec>( dg::create::weights( g1d));
    const dg::DVec w2d = dg::construct<dg::DVec>( dg::create::weights( g2d));
    const dg::fDVec wf2d = dg::construct<dg::fDVec>( dg::create::weights( gf2d));
    const dg::DVec w3d = dg::construct<dg::DVec>( dg::create::weights( g3d));
    exblas::udouble res;

    double integral = dg::blas1::dot( w1d, func1d); res.d = integral;
    std::cout << "1D integral               "<<std::setw(6)<<integral <<"\t" << res.i - 4616944842743393935  << "\n";
    double sol = (exp(2.) -exp(1));
    std::cout << "Correct integral is       "<<std::setw(6)<<sol<<std::endl;
    std::cout << "Relative 1d error is      "<<(integral-sol)/sol<<"\n\n";

    double integral2d = dg::blas1::dot( w2d, func2d); res.d = integral2d;
    std::cout << "2D integral               "<<std::setw(6)<<integral2d <<"\t" << res.i - 4639875759346476257<< "\n";
    double sol2d = (exp(2.)-exp(1))*(exp(4.)-exp(3));
    std::cout << "Correct integral is       "<<std::setw(6)<<sol2d<<std::endl;
    std::cout << "Relative 2d error is      "<<(integral2d-sol2d)/sol2d<<"\n\n";

    float integralf2d = dg::blas1::dot( wf2d, func2d); res.d = integralf2d;
    std::cout << "2D integral (float)       "<<std::setw(6)<<integralf2d <<"\t" << res.i - 4639875760323035136<< "\n";
    float solf2d = (exp(2.)-exp(1))*(exp(4.)-exp(3));
    std::cout << "Correct integral is       "<<std::setw(6)<<solf2d<<std::endl;
    std::cout << "Relative 2d error (float) "<<(integralf2d-solf2d)/solf2d<<"\n\n";

    double integral3d = dg::blas1::dot( w3d, func3d); res.d = integral3d;
    std::cout << "3D integral               "<<std::setw(6)<<integral3d <<"\t" << res.i - 4675882723962622631<< "\n";
    double sol3d = sol2d*(exp(6.)-exp(5.));
    std::cout << "Correct integral is       "<<std::setw(6)<<sol3d<<std::endl;
    std::cout << "Relative 3d error is      "<<(integral3d-sol3d)/sol3d<<"\n\n";

    double norm = dg::blas2::dot( func1d, w1d, func1d); res.d = norm;
    std::cout << "Square normalized 1D norm "<<std::setw(6)<<norm<<"\t" << res.i - 4627337306989890294 <<"\n";
    double solution = (exp(4.) -exp(2))/2.;
    std::cout << "Correct square norm is    "<<std::setw(6)<<solution<<std::endl;
    std::cout << "Relative 1d error is      "<<(norm-solution)/solution<<"\n\n";

    double norm2d = dg::blas2::dot( w2d, func2d); res.d = norm2d;
    std::cout << "Square normalized 2D norm "<<std::setw(6)<<norm2d<<"\t" << res.i - 4674091193523851724<<"\n";
    double solution2d = (exp(4.)-exp(2))/2.*(exp(8.) -exp(6.))/2.;
    std::cout << "Correct square norm is    "<<std::setw(6)<<solution2d<<std::endl;
    std::cout << "Relative 2d error is      "<<(norm2d-solution2d)/solution2d<<"\n\n";

    double norm3d = dg::blas2::dot( func3d, w3d, func3d); res.d = norm3d;
    std::cout << "Square normalized 3D norm "<<std::setw(6)<<norm3d<<"\t" << res.i - 4746764681002108278<<"\n";
    double solution3d = solution2d*(exp(12.) -exp(10.))/2.;
    std::cout << "Correct square norm is    "<<std::setw(6)<<solution3d<<std::endl;
    std::cout << "Relative 3d error is      "<<(norm3d-solution3d)/solution3d<<"\n\n";

    std::cout << "TEST result of a sin and exp function to compare compiler specific math libraries:\n";
    dg::DVec x(1, 6.12610567450009658);
    dg::blas1::transform( x, x, sin_function() );
    res.d = x[0];
    std::cout << "Result of sin:    "<<res.i<<"\n"
              << "          GCC:    -4628567870976535683 (correct)"<<std::endl;
    dg::DVec y(1, 5.9126151457310376);
    dg::blas1::transform( y, y, exp_function() );
    res.d = y[0];
    std::cout << "Result of exp:     "<<res.i<<"\n"
              << "          GCC:     4645210948416067678 (correct)"<<std::endl;

    //TEST OF INTEGRAL
    dg::HVec integral_num = dg::integrate( cos, g1d);
    dg::HVec integral_ana = dg::evaluate( sin, g1d);
    dg::blas1::plus( integral_ana, -sin(g1d.x0()));
    dg::blas1::axpby( 1., integral_ana, -1., integral_num);
    norm = dg::blas2::dot( integral_num, dg::create::weights( g1d), integral_num);
    std::cout << " Error norm of  1d integral function "<<norm<<"\n";
    // TEST if dot throws on NaN
    dg::blas1::transform( x,x, dg::LN<double>());
    try{
        dg::blas1::dot( x,x);
    }catch ( std::exception& e)
    {
        std::cerr << "Error thrown as expected\n";
        std::cerr << e.what() << std::endl;
    }

    std::cout << "\nFINISHED! Continue with topology/derivatives_t.cu !\n\n";
    return 0;
}
